#include "hip/hip_runtime.h"
#include "Header.h"

__global__ void add_function_GPU(float* a, float* b, float* c, int N)
{
	// ���ڵ�ǰ�߳����ڵ�����飬���������������е�����
	int block_id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

	// ���ڵ�ǰ�̣߳���������������������߳��������Լ���ǰ�߳��ڵ�ǰ������е�����
	int thread_id = (block_id * blockDim.x * blockDim.y * blockDim.z)
		+ (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y);

	if (thread_id < N)
	{
		c[thread_id] = a[thread_id] + b[thread_id];

		//printf("Block(%d %d %d) = %d || Thread(%d %d %d) = %d %f %f %f\n", blockIdx.x, blockIdx.y, blockIdx.z, block_id,
		//	threadIdx.x, threadIdx.y, threadIdx.z, thread_id, a[thread_id], b[thread_id], c[thread_id]);
	}
}