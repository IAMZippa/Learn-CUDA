﻿#include "Header.h"

// 版本：Visual Studio 2019 + CUDA 11.8

int main()
{
	int N = 100000000; // size of float 

	// 1.1 alloc memory in CPU
	float* h_a = (float*)malloc(N * sizeof(float));
	float* h_b = (float*)malloc(N * sizeof(float));
	float* h_c = (float*)malloc(N * sizeof(float));

	// 1.2 initialize memory in CPU
	for (int i = 0; i < N; i++)
	{
		h_a[i] = i;
		h_b[i] = i;
	}

	// 2.1 alloc memory in GPU
	float* d_a, *d_b, *d_c;
	hipMalloc(&d_a, N * sizeof(float)); // 注意这个语法，不要用错了
	hipMalloc(&d_b, N * sizeof(float));
	hipMalloc(&d_c, N * sizeof(float));

	// 2.2 copy memory from CPU to GPU
	hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, N * sizeof(float), hipMemcpyHostToDevice);

	// 3 run function in GPU
	int blockPerGrid = (N + BlockSize - 1) / BlockSize;

	add_function_GPU<<<blockPerGrid, BlockSize >>> (d_a, d_b, d_c, N);
	hipDeviceSynchronize(); // do remember to Synchronize

	// 3.1 copy memory from GPU to CPU
	hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

	// 3.2 Compare GPU and CPU Results
	for (int i = 0; i < N; i++)
	{
		int h_c_CPU = h_a[i] + h_b[i];
		if (h_c[i] != h_c_CPU)
		{
			printf("Error!");
		}
	}
	printf("Same results between CPU and GPU\n");

	// let's see the running time between CPU and GPU
	{
		auto start = std::chrono::high_resolution_clock::now();  // 开始计时

		for (int istep = 0; istep < 20; istep++)
		{
			add_function_CPU(h_a, h_b, h_c, N);
		}

		auto end = std::chrono::high_resolution_clock::now();  // 结束计时
		std::chrono::duration<double> elapsed = end - start;   // 计算时间差
		// 输出当前步的执行时间
		printf("CPU exeTime %f seconds \n", elapsed.count());
	}

	{
		auto start = std::chrono::high_resolution_clock::now();  // 开始计时

		for (int istep = 0; istep < 20; istep++)
		{
			add_function_GPU << <blockPerGrid, BlockSize >> > (d_a, d_b, d_c, N);
			hipDeviceSynchronize();
		}

		auto end = std::chrono::high_resolution_clock::now();  // 结束计时
		std::chrono::duration<double> elapsed = end - start;   // 计算时间差
		// 输出当前步的执行时间
		printf("GPU exeTime %f seconds \n", elapsed.count());
	}

	// 4
	hipFree(d_a);
	hipFree(d_a);
	hipFree(d_a);

	free(h_a);
	free(h_b);
	free(h_c);
}


