﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>


void addfunction(int N, float* h_a, float* h_b, float* h_c)
{
	for (int i = 0; i < N; i++)
	{
		h_c[i] = h_a[i] + h_b[i];
	}
}

int main()
{
	// 1 memory alloc in CPU
	int N = 5;
	float* h_a = new float[N]();
	float* h_b = new float[N]();
	float* h_c = new float[N]();

	// initialize 
	for (int i = 0; i < N; i++)
	{
		h_a[i] = i;
		h_b[i] = i * 2;
		printf("%d %f %f %f\n", i, h_a[i], h_b[i], h_c[i]);
	}





}

